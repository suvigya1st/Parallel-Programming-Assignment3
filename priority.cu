
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define THREADS 256 
#define BLOCKS 32 
#define NUM THREADS*BLOCKS

int seed_var =1239;

int random_int()
{
  return (int)rand()%(int)9 +1;
}

void array_fill(int *arr, int length)
{
  srand(++seed_var);
  int i;
  for (i = 0; i < length; ++i) {
    arr[i] = random_int();
  }
}

void print_array(int *arr1,int *arr2 ,int *arr3,int *arr4, int length)
{
  //srand(time(NULL));
  int i;
  for (i = 0; i < length; ++i) {
    //arr[i] = random_float();
    printf("%d\t%d\t%d\t%d\t%d\n",i+1,arr1[i],arr2[i],arr3[i],arr4[i]);
  }
}

void print_elapsed(clock_t start, clock_t stop)
{
  double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
  printf("Elapsed time: %.3fs for %d process\n", elapsed, NUM);
}

__device__ void swap(int *xp, int *yp)
{
    int temp = *xp;
    *xp = *yp;
    *yp = temp;
}
__global__ void bitonic_sort_step(int *d_pr, int *d_bt, int j, int k)
{
  int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) 
  {
    if ((i&k)==0) 
    {
      /* Sort ascending */
      if (d_pr[i]>d_pr[ixj]) 
      {
        /* exchange(i,ixj); */
        swap(&d_pr[i],&d_pr[ixj]);
        swap(&d_bt[i],&d_bt[ixj]);
      }
    }
    if ((i&k)!=0)
    {
      /* Sort descending */
      if (d_pr[i]<d_pr[ixj])
      {
        /* exchange(i,ixj); */
        swap(&d_pr[i], &d_pr[ixj]);
        swap(&d_bt[i], &d_bt[ixj]);
      }
    }
  }
}

void sorting_first(int *pr, int *bt)
{
  
  dim3 blocks(BLOCKS,1);
  dim3 threads(THREADS,1);

  int k;
  
  //Major step priority time basis sorting
  for(k = 2; k <= NUM; k <<= 1)
  {
    for (int j = k>>1; j > 0; j = j>>1)
    {
      bitonic_sort_step<<<blocks,threads>>>(pr,bt,j,k);
    }
  }
}



__global__ void work_efficient_scan_kernel(int *X, int *Y, int InputSize)
{
  extern __shared__ int XY[];
  int i= blockIdx.x*blockDim.x+ threadIdx.x;
  if (i < InputSize)
  {
    XY[threadIdx.x] = X[i];
  }
  for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
  {
    __syncthreads();
    int index = (threadIdx.x+1) * 2* stride -1;
    if (index < blockDim.x)
    {
      XY[index] += XY[index -stride];
    }
  }
  for (int stride = THREADS/4; stride > 0; stride /= 2)
  {
    __syncthreads();
    int index = (threadIdx.x+1)*stride*2 -1;
    if(index + stride < THREADS)
    {
     XY[index + stride] += XY[index];
    }
  }
  __syncthreads();
  Y[i] = XY[threadIdx.x];

  //OWN CODE
  __syncthreads();
  if(threadIdx.x < blockIdx.x)
  {
    XY[threadIdx.x] = Y[threadIdx.x*blockDim.x + (blockDim.x-1)];
  }
  __syncthreads();
  for(unsigned int stride =0; stride < blockIdx.x; stride++)
  {
    Y[i] += XY[stride];
  }
  __syncthreads();
}

void scan_next(int *bt, int *tat)
{
  dim3 blocks(BLOCKS,1);
  dim3 threads(THREADS,1);

  work_efficient_scan_kernel<<<blocks, threads, THREADS * sizeof(int)>>>(bt, tat,NUM);
}

int main()
{
  
  clock_t start, stop;
  int *h_bt = (int*) malloc( NUM * sizeof(int));
  int *h_pr = (int*) malloc( NUM * sizeof(int));
  int *h_tat = (int*) malloc( NUM * sizeof(int));
  int *h_wt = (int*) malloc( NUM * sizeof(int));
  array_fill(h_bt, NUM);
  array_fill(h_pr, NUM);
  printf("INITIAL\n");
  printf("\tPR\tBT\tWT\tTAT\n");
  print_array(h_pr,h_bt,h_wt,h_tat,NUM);

  int *d_bt, *d_pr, *d_wt, *d_tat;
  size_t size = NUM * sizeof(int);

  hipMalloc((void**) &d_bt, size);
  hipMalloc((void**) &d_pr, size);
  hipMalloc((void**) &d_wt, size);
  hipMalloc((void**) &d_tat, size);

  hipMemcpy(d_bt, h_bt, size, hipMemcpyHostToDevice);
  hipMemcpy(d_pr, h_pr, size, hipMemcpyHostToDevice);


  start = clock();
  sorting_first(d_pr, d_bt);
  
  scan_next(d_bt, d_tat);
  
  hipMemcpy(h_bt, d_bt, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_pr, d_pr, size, hipMemcpyDeviceToHost);
  hipMemcpy(&h_wt[1], d_tat, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_tat, d_tat, size, hipMemcpyDeviceToHost);

  hipFree(d_pr);
  hipFree(d_bt);
  hipFree(d_wt);
  hipFree(d_tat);
  
  stop = clock();

  printf("\nFINAL\n");
  printf("\tPR\tBT\tWT\tTAT\n");
  print_array(h_pr,h_bt,h_wt,h_tat,NUM);
  print_elapsed(start, stop);

} 